#include "hip/hip_runtime.h"
#include <iostream>


__global__ void VecAdd(float* x1, float* x2, float* y)
{
    int i  = threadIdx.x;
    y[i] = x1[i] + x2[i];
}

void main()
{
    float* x1 = hipMalloc(1024*sizeof(float));
    float* x2 = hipMalloc(1024*sizeof(float));
    float* y  = hipMalloc(1024*sizeof(float));

    VecAdd<<<1, 1024>(x1, x2, y);

    // Free memory.
    hipFree(x1); x1 = nullptr;
    hipFree(x2); x2 = nullptr;
    hipFree(y);   y = nullptr;
}