
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <vector>
#include <cassert>


// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if(err != hipSuccess) { \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)


template<typename T>
std::shared_ptr<T> CreateTensor(size_t N)
{
    T * data;
    CUDA_CHECK(hipMalloc(&data, N*sizeof(T)));
    return std::shared_ptr<T>(data, [](T* data){ 
        CUDA_CHECK(hipFree(data));
    });
}

template<typename T, size_t N>
struct CudaVector
{
    std::array<T, N> host;
    void ToDevice()
    {
        mDevice = CreateTensor<T>(host.size());
        CUDA_CHECK(hipMemcpy(mDevice.get(), host.data(), sizeof(T)*N, hipMemcpyHostToDevice));
    }

    void ToHost()
    {
        CUDA_CHECK(hipMemcpy(host.data(), mDevice.get(), sizeof(T)*N, hipMemcpyDeviceToHost));
    }

    T * device_ptr() { return mDevice.get(); };


    std::shared_ptr<T> mDevice;
};

__global__ void VecAdd(float* x1, float* x2, float* y)
{
    int i  = threadIdx.x;
    y[i] = 100.0f;//x1[i] + x2[i];
}

int main()
{
    CudaVector<float, 1024> x1;
    CudaVector<float, 1024> x2;
    CudaVector<float, 1024> y;

    for(int i = 0 ; i < x1.host.size(); ++i)
    {
        x1.host[i] = i*1.0f;
        x2.host[i] = 1024.0f - i*1.0f;
        y.host[i]  = 0.0f;
    }

    x1.ToDevice();
    x2.ToDevice();
    y.ToDevice();

    VecAdd<<<1, 1024>>>(x1.device_ptr(), x2.device_ptr(), y.device_ptr());
    CUDA_CHECK(hipDeviceSynchronize());
    y.ToHost();

    for(int i = 0 ; i < x1.host.size(); ++i)
    {
        std::cout << x1.host[i] << " + " << x2.host[i] << " =" << y.host[i] << std::endl;
    }

    return 0;
}